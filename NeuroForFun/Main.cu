#include "hip/hip_runtime.h"
﻿#include "definitions.h"
//#include "Mtrx.cu.h"
#include "HostMtrx.h"
#include "TrainSet.h"
#include "Neuro.h"

template <typename T>
T random(T low, T high) {
	thread_local static std::random_device rd;
	thread_local static std::mt19937 rng(rd());
	thread_local std::uniform_real_distribution<> urd;
	return urd(rng, decltype(urd)::param_type{low,high});
}

template <typename T>
T func(T x) {
	return sin(x);
}

template <typename T>
bool eq(const T a, const T b, const T epsilon) {
	if(abs(a / b - 1) < epsilon)
		return true;
	return false;
}

template <typename T>
int testGpuOnCpuMtrx(const T epsilon) {
	MtrxFactory<T> *gpuMtrxFactory = new GPUMtrxFactory<T>;
	MtrxFactory<T> *cpuMtrxFactory = new CPUMtrxFactory<T>;

	int m = 2,
		k = 3,
		n = 2;

	std::vector<Mtrx<T> *> gm;
	gm.push_back(gpuMtrxFactory->create(m, k));
	gm.push_back(gpuMtrxFactory->create(k, n));
	gm.push_back(gpuMtrxFactory->create(m, n, CLEAR));
	gm.push_back(gpuMtrxFactory->create(m, n));

	std::vector<Mtrx<T> *> cm;
	cm.push_back(cpuMtrxFactory->create(m, k, CLEAR));
	cm.push_back(cpuMtrxFactory->create(k, n, CLEAR));
	cm.push_back(cpuMtrxFactory->create(m, n, CLEAR));
	cm.push_back(cpuMtrxFactory->create(m, n, CLEAR));
	FOR(k, 4)
		FOR(i, gm[k]->getH())
			FOR(j, gm[k]->getW())
				cm[k]->set(i * gm[k]->getW() + j, gm[k]->get(i, j));

	// activation
	{
		gm[2]->activation();
		cm[2]->activation();

		FOR(i, gm[2]->getH())
			FOR(j, gm[2]->getW())
			if(!eq<T>(cm[2]->get(i, j), gm[2]->get(i, j), epsilon)) return -1;

		FOR(k, 4)
			FOR(i, gm[k]->getH())
			FOR(j, gm[k]->getW())
			cm[k]->set(i * gm[k]->getW() + j, gm[k]->get(i, j));
	}

	// dActivation
	{
		gm[2]->dActivation();
		cm[2]->dActivation();

		FOR(i, gm[2]->getH())
			FOR(j, gm[2]->getW())
			if(!eq<T>(cm[2]->get(i, j), gm[2]->get(i, j), epsilon)) return -2;
		FOR(k, 4)
			FOR(i, gm[k]->getH())
			FOR(j, gm[k]->getW())
			cm[k]->set(i * gm[k]->getW() + j, gm[k]->get(i, j));
	}

	// backActivation
	{
		gm[2]->backActivation();
		cm[2]->backActivation();

		FOR(i, gm[2]->getH())
			FOR(j, gm[2]->getW())
			if(!eq<T>(cm[2]->get(i, j), gm[2]->get(i, j), epsilon)) return -3;
		FOR(k, 4)
			FOR(i, gm[k]->getH())
			FOR(j, gm[k]->getW())
			cm[k]->set(i * gm[k]->getW() + j, gm[k]->get(i, j));
	}

	// mult
	{
		gm[2]->mult(gm[0], gm[1]);
		cm[2]->mult(cm[0], cm[1]);

		FOR(i, gm[2]->getH())
			FOR(j, gm[2]->getW())
			if(!eq<T>(cm[2]->get(i, j), gm[2]->get(i, j), epsilon)) return -4;
		FOR(k, 4)
			FOR(i, gm[k]->getH())
			FOR(j, gm[k]->getW())
			cm[k]->set(i * gm[k]->getW() + j, gm[k]->get(i, j));
	}

	// lineMult
	{
		gm[2]->lineMult(gm[3]);
		cm[2]->lineMult(cm[3]);

		FOR(i, gm[2]->getH())
			FOR(j, gm[2]->getW())
			if(!eq<T>(cm[2]->get(i, j), gm[2]->get(i, j), epsilon)) return -5;
		FOR(k, 4)
			FOR(i, gm[k]->getH())
			FOR(j, gm[k]->getW())
			cm[k]->set(i * gm[k]->getW() + j, gm[k]->get(i, j));
	}

	// coeffMult
	{
		gm[2]->coeffMult(5);
		cm[2]->coeffMult(5);

		FOR(i, gm[2]->getH())
			FOR(j, gm[2]->getW())
			if(!eq<T>(cm[2]->get(i, j), gm[2]->get(i, j), epsilon)) return -6;
		FOR(k, 4)
			FOR(i, gm[k]->getH())
			FOR(j, gm[k]->getW())
			cm[k]->set(i * gm[k]->getW() + j, gm[k]->get(i, j));
	}

	// lineSub
	{
		gm[2]->lineSub(gm[3]);
		cm[2]->lineSub(cm[3]);

		FOR(i, gm[2]->getH())
			FOR(j, gm[2]->getW())
			if(!eq<T>(cm[2]->get(i, j), gm[2]->get(i, j), epsilon)) return -7;
		FOR(k, 4)
			FOR(i, gm[k]->getH())
			FOR(j, gm[k]->getW())
			cm[k]->set(i * gm[k]->getW() + j, gm[k]->get(i, j));
	}

	// transpose
	{
		gm[1]->transpose();
		cm[1]->transpose();

		FOR(i, gm[1]->getH())
			FOR(j, gm[1]->getW())
			if(!eq<T>(cm[1]->get(i, j), gm[1]->get(i, j), epsilon)) return -8;

		gm[1]->transpose();
		cm[1]->transpose();

		FOR(i, gm[1]->getH())
			FOR(j, gm[1]->getW())
			if(!eq<T>(cm[1]->get(i, j), gm[1]->get(i, j), epsilon)) return -9;
		FOR(k, 4)
			FOR(i, gm[k]->getH())
			FOR(j, gm[k]->getW())
			cm[k]->set(i * gm[k]->getW() + j, gm[k]->get(i, j));
	}

	// setKernel
	{
		FOR(i, 4) {
			gm[2]->set(i, i+.2f);
			cm[2]->set(i, i+.2f);
		}

		FOR(i, gm[2]->getH())
			FOR(j, gm[2]->getW())
			if(!eq<T>(cm[2]->get(i, j), gm[2]->get(i, j), epsilon)) return -10;
		FOR(k, 4)
			FOR(i, gm[k]->getH())
			FOR(j, gm[k]->getW())
			cm[k]->set(i * gm[k]->getW() + j, gm[k]->get(i, j));
	}

	return 1;
}

int main() {
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if(cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
	}

	// https://en.wikipedia.org/wiki/Machine_epsilon#Values_for_standard_hardware_floating_point_arithmetics
	std::cout << "test code: " << testGpuOnCpuMtrx<float>(1.19e-07f) << "\r\n";

	/*float trIn[TRAINSET_SIZE * INPUT_SIZE];
	float trOut[TRAINSET_SIZE * OUTPUT_SIZE];
	float x = 0, maxX = std::_Pi, num = 20, stepX = (maxX - x) / (num - 1);
	int epochs = 1000, cols = 10;
	FOR(i, TRAINSET_SIZE) {
		trIn[i] = x;
		trOut[i] = func<float>(x);
		x += stepX;
	}
	int *layersSizes = new int[LAYERS_COUNT] { INPUT_SIZE, 10, 10, 10, 10, OUTPUT_SIZE };
	std::vector<TrainSet<float> *> trainsets;
	trainsets.push_back(new TrainSet<float>(trIn, trOut, TRAINSET_SIZE, INPUT_SIZE, OUTPUT_SIZE));

	auto t1 = std::chrono::high_resolution_clock::now();
	MtrxFactory<float> *gpuMtrxFactory = new GPUMtrxFactory<float>;
	Neuro<float> *INN = new Neuro<float>(layersSizes, trainsets[0], gpuMtrxFactory);
	float *MSE = new float[cols];
	FOR(i, cols) {
		FOR(j, epochs)
			INN->train(0.01f);
		MSE[i] = INN->MSE;
	}
	auto t2 = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count();
	FOR(i, cols) {
		std::cout << MSE[i] << "\r\n";
	}
	std::cout << "Done in " << duration / 1e6 << " seconds." << std::endl;
	/**/

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	if(cuSafe(hipDeviceReset(), "hipDeviceReset failed!")) return -1;

	return 0;
}